#include "hip/hip_runtime.h"
//
// Created by pierfied on 10/5/20.
//

#include <healpix_cxx/healpix_base.h>
#include <iostream>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>

#include "standard_transforms.cuh"
#include "wigner.cuh"

#define CHUNKSIZE 16

__global__ void FKernel(int lmax, int nrings, int nchunks, hipDoubleComplex *F, hipDoubleComplex *alm,
                        double *ringTheta, double *ringPhi0, double *fac1, double *fac2, double *fac3) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < (lmax + 1) * nchunks; i += stride) {
        int m = i / nchunks;
        int c = i % nchunks;
        int yOffset = c * CHUNKSIZE;

        hipDoubleComplex FVals[CHUNKSIZE];
        double dlm[CHUNKSIZE], prevDlm[CHUNKSIZE], cosTheta[CHUNKSIZE], cosPhase[CHUNKSIZE], sinPhase[CHUNKSIZE];

#pragma unroll
        for (int j = 0; j < CHUNKSIZE; j++) {
            dlm[j] = emmRecursionSeed(m, 0, ringTheta[yOffset + j]);
            prevDlm[j] = 0;
            cosTheta[j] = cos(ringTheta[yOffset + j]);
            cosPhase[j] = cos(ringPhi0[yOffset + j]);
            sinPhase[j] = sin(ringPhi0[yOffset + j]);
            FVals[j].x = 0;
            FVals[j].y = 0;
        }

        int ind = m * (lmax + 1) - (m - 1) * m / 2;
        double sLambdalm[CHUNKSIZE], tmpDlm[CHUNKSIZE];

        for (int l = m; l <= lmax; l++){
            double prefac = pow(-1, m) * sqrt((2. * l + 1) / (4 * M_PI));
            hipDoubleComplex almVal = alm[ind];
            double fac1Val = fac1[ind];
            double fac3Val = fac3[ind];

//#pragma unroll
//            for (int j = 0; j < CHUNKSIZE; j++){
//                sLambdalm[j] = prefac * dlm[j];
//            }

#pragma unroll
            for (int j = 0; j < CHUNKSIZE; j++){
                sLambdalm[j] = prefac * dlm[j];
                tmpDlm[j] = fac1Val * cosTheta[j] * dlm[j] - fac3Val * prevDlm[j];
                prevDlm[j] = dlm[j];
                dlm[j] = tmpDlm[j];
                FVals[j].x += almVal.x * sLambdalm[j];
                FVals[j].y += almVal.y * sLambdalm[j];
            }

//#pragma unroll
//            for (int j = 0; j < CHUNKSIZE; j++){
//                tmpDlm[j] = fac1Val * cosTheta[j] * dlm[j] - fac3Val * prevDlm[j];
//                prevDlm[j] = dlm[j];
//                dlm[j] = tmpDlm[j];
//            }

//#pragma unroll
//            for (int j = 0; j < CHUNKSIZE; j++){
//            }

            ind++;
        }

#pragma unroll
        for (int j = 0; j < CHUNKSIZE; j++){
            F[(yOffset + j) * (lmax + 1) + m] = hipCmul(FVals[j], make_hipDoubleComplex(cosPhase[j], sinPhase[j]));
        }
    }
}

torch::Tensor alm2map(torch::Tensor alm, int nside, int lmax) {
    // Start computing the recursion coefficients on the GPU now while we compute Healpix stuff next.
    double *fac1, *fac2, *fac3;
    computeRecursionCoeffs(lmax, 0, &fac1, &fac2, &fac3);

    // Create the base Healpix class for useful routines later.
    nside_dummy dummy;
    Healpix_Base base(nside, RING, dummy);

    // Compute the size of the map and number of rings.
    int npix = 12 * nside * nside;
    int nrings = base.pix2ring(npix - 1);
    int nchunks = (nrings + CHUNKSIZE - 1) / CHUNKSIZE;
    int nringsPad = nchunks * CHUNKSIZE;

    // Create the CUDA arrays for the ring info.
    int *ringPix, *ringStart;
    double *ringTheta, *ringPhi0;
    hipMallocManaged(&ringPix, sizeof(int) * nringsPad);
    hipMallocManaged(&ringStart, sizeof(int) * nringsPad);
    hipMallocManaged(&ringTheta, sizeof(double) * nringsPad);
    hipMallocManaged(&ringPhi0, sizeof(double) * nringsPad);

    // Get all of the relevant info for the ring.
#pragma omp parallel for
    for (int i = 0; i < nrings; i++) {
        bool shifted;
        base.get_ring_info2(i + 1, ringStart[i], ringPix[i], ringTheta[i], shifted);
        ringPhi0[i] = base.pix2ang(ringStart[i]).phi;
    }

    // Create the F array and get the pointer to the alm data.
    hipDoubleComplex *almPtr, *F;
    almPtr = (hipDoubleComplex *) alm.data<torch::complex<double>>();
    hipMallocManaged(&F, sizeof(hipDoubleComplex) * (lmax + 1) * nringsPad);

    // Launch the kernel to compute F.
    int blockSize, gridSize;
    hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, FKernel, 0, 0);
    FKernel<<<gridSize, blockSize>>>(lmax, nrings, nchunks, F, almPtr, ringTheta, ringPhi0, fac1, fac2, fac3);

    // Create the map tensor.
    torch::Tensor map = torch::zeros(npix, torch::dtype(torch::kFloat64).device(torch::kCUDA));
    double *mapPtr = map.data<double>();

    // Perform the FFTs to build the map.
    for (int i = 0; i < nrings; i++) {
        hipfftHandle plan;
        double *ringPtr = &mapPtr[ringStart[i]];
        hipfftDoubleComplex *data = &F[i * (lmax + 1)];
        hipfftPlan1d(&plan, ringPix[i], HIPFFT_Z2D, 1);
        hipfftExecZ2D(plan, data, ringPtr);
        hipfftDestroy(plan);
    }

    // Free arrays.
    hipFree(fac1);
    hipFree(fac2);
    hipFree(fac3);
    hipFree(ringPix);
    hipFree(ringStart);
    hipFree(ringTheta);
    hipFree(ringPhi0);
    hipFree(F);

    return map;
}
